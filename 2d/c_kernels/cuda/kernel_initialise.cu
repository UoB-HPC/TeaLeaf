#include "hip/hip_runtime.h"
#include "../../shared.h"
#include "c_kernels.h"
#include "cuknl_shared.h"
#include <algorithm>
#include <iostream>
#include <cstdlib>
#include <string>
#include <vector>

// Allocates, and zeroes and individual buffer
void allocate_device_buffer(double** a, int x, int y)
{

    hipMalloc((void**)a, x*y*sizeof(double));
    check_errors(__LINE__, __FILE__);

    int num_blocks = ceil((double)(x*y)/(double)BLOCK_SIZE);
    zero_buffer<<<num_blocks, BLOCK_SIZE>>>(x, y, *a);
    check_errors(__LINE__, __FILE__);
}

void allocate_host_buffer(double** a, int x, int y)
{
    *a = (double*)malloc(sizeof(double)*x*y);

    if(*a == NULL) 
    {
        die(__LINE__, __FILE__, "Error allocating buffer %s\n");
    }

#pragma omp parallel for
    for(int jj = 0; jj < y; ++jj)
    {
        for(int kk = 0; kk < x; ++kk)
        {
            const int index = kk + jj*x;
            (*a)[index] = 0.0;
        }
    }
}

// Allocates all of the field buffers
void kernel_initialise(
        Settings* settings, int x, int y, double** density0, 
        double** density, double** energy0, double** energy, double** u, 
        double** u0, double** p, double** r, double** mi, 
        double** w, double** kx, double** ky, double** sd, 
        double** volume, double** x_area, double** y_area, double** cell_x, 
        double** cell_y, double** cell_dx, double** cell_dy, double** vertex_dx, 
        double** vertex_dy, double** vertex_x, double** vertex_y,
        double** cg_alphas, double** cg_betas, double** cheby_alphas,
        double** cheby_betas, double** d_comm_buffer, double** d_reduce_buffer, 
        double** d_reduce_buffer2, double** d_reduce_buffer3, double** d_reduce_buffer4)
{

    print_and_log(settings,
                  "Performing this solve with the CUDA %s solver\n",
                  settings->solver_name);

    int count;
    hipGetDeviceCount(&count);
    std::vector<std::pair<int, std::string>> devices(count);
    for (int i = 0; i < count; ++i) {
        hipDeviceProp_t props{};
        hipGetDeviceProperties(&props, i);
        devices[i] = {i, std::string(props.name)};
    }

    print_and_log(settings, "Available devices = %d\n", devices.size());
    if(count == 0) {
        print_and_log(settings, "WARNING: hipGetDeviceCount returned 0 devices.\n");
    }
    for(auto &d : devices) {
        print_and_log(settings, "\t[%d] %s\n", d.first, d.second.c_str());
    }

    auto selector = !settings->device_selector ? "0" : std::string(settings->device_selector);
    int selected = 0;
    try {
        selected = std::stoi(selector);
    } catch (const std::exception &e) {
        print_and_log(settings, "Unable to parse/select device index `%s`: %s\n", selector.c_str(), e.what());
        print_and_log(settings, "Attempting to match device with substring  `%s`\n", selector.c_str());

        auto matching = std::find_if(devices.begin(), devices.end(),
                                     [selector](const auto &device) { return device.second.find(selector) != std::string::npos; });
        if (matching != devices.end()) {
            selected = matching->first;
            print_and_log(settings, "Using first device matching substring `%s`\n", selector.c_str());
        } else if (devices.size() == 1)
            print_and_log(settings, "No matching device but there's only one device, will be using that anyway\n");
        else {
            die(__LINE__, __FILE__, "No matching devices for `%s`\n", selector.c_str());
        }
    }

    int result = hipSetDevice(selected);
    if(result != hipSuccess)
    {
        die(__LINE__,__FILE__,"Could not allocate CUDA device %d.\n", selected);
    }

    hipDeviceProp_t properties{};
    hipGetDeviceProperties(&properties, selected);
    print_and_log(settings, "Rank %d using %s device id %d\n", settings->rank, properties.name, selected);

    const int x_inner = x - 2*settings->halo_depth;
    const int y_inner = y - 2*settings->halo_depth;

    allocate_device_buffer(density0, x, y);
    allocate_device_buffer(density, x, y);
    allocate_device_buffer(energy0, x, y);
    allocate_device_buffer(energy, x, y);
    allocate_device_buffer(u, x, y);
    allocate_device_buffer(u0, x, y);
    allocate_device_buffer(p, x, y);
    allocate_device_buffer(r, x, y);
    allocate_device_buffer(mi, x, y);
    allocate_device_buffer(w, x, y);
    allocate_device_buffer(kx, x, y);
    allocate_device_buffer(ky, x, y);
    allocate_device_buffer(sd, x, y);
    allocate_device_buffer(volume, x, y);
    allocate_device_buffer(x_area, x+1, y);
    allocate_device_buffer(y_area, x, y+1);
    allocate_device_buffer(cell_x, x, 1);
    allocate_device_buffer(cell_y, 1, y);
    allocate_device_buffer(cell_dx, x, 1);
    allocate_device_buffer(cell_dy, 1, y);
    allocate_device_buffer(vertex_dx, x+1, 1);
    allocate_device_buffer(vertex_dy, 1, y+1);
    allocate_device_buffer(vertex_x, x+1, 1);
    allocate_device_buffer(vertex_y, 1, y+1);
    allocate_device_buffer(d_comm_buffer, settings->halo_depth, max(x_inner, y_inner));
    allocate_device_buffer(d_reduce_buffer, x, y);
    allocate_device_buffer(d_reduce_buffer2, x, y);
    allocate_device_buffer(d_reduce_buffer3, x, y);
    allocate_device_buffer(d_reduce_buffer4, x, y);

    allocate_host_buffer(cg_alphas, settings->max_iters, 1);
    allocate_host_buffer(cg_betas, settings->max_iters, 1);
    allocate_host_buffer(cheby_alphas, settings->max_iters, 1);
    allocate_host_buffer(cheby_betas, settings->max_iters, 1);
}

// Finalises the kernel
void kernel_finalise(
        double* cg_alphas, double* cg_betas, double* cheby_alphas,
        double* cheby_betas)
{
    free(cg_alphas);
    free(cg_betas);
    free(cheby_alphas);
    free(cheby_betas);
}
